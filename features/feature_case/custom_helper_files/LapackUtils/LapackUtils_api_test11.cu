// ===------ LapackUtils_api_test11.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_gesvd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  float* s_s;
  float* u_s;
  float* vt_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s, 2, HIP_R_32F, s_s, HIP_R_32F, u_s, 2, HIP_R_32F, vt_s, 2, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}